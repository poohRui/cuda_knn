#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

/**
 * This is a stub function which call gemm to calculate "matrix * matrix" problem
 *
 *@param A    an input matrix with size m*dim
 *@param B    an input matrix with size dim*n
 *@param C    an output matrix with size m*n
 *@param m    size of the first dimension in A
 *@param n    size of the second dimension in B
 *@param dim  size of the second dimension in A
 */
void gemm_cuda(float* A,
               float* B,
               float* C,
               int    m,
               int    n,
               int    dim){
    
    float* d_A;
    float* d_B;
    float* d_C;
    
    size_t size_of_float = sizeof(float);
    size_t size_pitch_bytes_A;
    size_t size_pitch_bytes_B;
    size_t size_pitch_bytes_C;
    
    // A(m*dim), B(dim*n), C(m*n)
    // Allocate space and transfer data to device
    hipMallocPitch((void**)&d_A, &size_pitch_bytes_A, dim * size_of_float, m);
    hipMallocPitch((void**)&d_B, &size_pitch_bytes_B, n * size_of_float, dim);
    hipMallocPitch((void**)&d_C, &size_pitch_bytes_C, n * size_of_float, m);
    
    hipMemcpy2D(d_A, size_pitch_bytes_A, A, dim * size_of_float, dim * size_of_float, m, hipMemcpyHostToDevice);
    hipMemcpy2D(d_B, size_pitch_bytes_B, B, n * size_of_float, n * size_of_float, dim, hipMemcpyHostToDevice);
    
    // Call gemm function in cublas
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    
    // Pay attention to the parameters
    float alpha = 1.0;
    float beta = 0.0;
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, dim, &alpha, d_B, n, d_A, dim, &beta, d_C, n);
    
    // Transfer result back
    hipMemcpy2D(C, n * size_of_float, d_C, size_pitch_bytes_C, n * size_of_float, m, hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    hipblasDestroy(cublasHandle);
}
