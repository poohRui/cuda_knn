#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"


void gemm_cuda(float* A,
               float* B,
               float* C,
               int    m,
               int    n,
               int    dim){
    
    float* d_A;
    float* d_B;
    float* d_C;
    
    size_t size_of_float = sizeof(float);
    size_t size_pitch_bytes_A;
    size_t size_pitch_bytes_B;
    size_t size_pitch_bytes_C;
    
    // A(m*dim), B(dim*n), C(m*n)
    hipMallocPitch((void**)&d_A, &size_pitch_bytes_A, dim * size_of_float, m);
    hipMallocPitch((void**)&d_B, &size_pitch_bytes_B, n * size_of_float, dim);
    hipMallocPitch((void**)&d_C, &size_pitch_bytes_C, n * size_of_float, m);
    
    hipMemcpy2D(d_A, size_pitch_bytes_A, A, dim * size_of_float, dim * size_of_float, m, hipMemcpyHostToDevice);
    hipMemcpy2D(d_B, size_pitch_bytes_B, B, n * size_of_float, n * size_of_float, dim, hipMemcpyHostToDevice);
    
    // 调用cublas的gemm的kernel，做矩阵乘矩阵的计算
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    
    float alpha = 1.0;
    float beta = 0.0;
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, dim, &alpha, d_B, n, d_A, dim, &beta, d_C, n);
    
    hipMemcpy2D(C, n * size_of_float, d_C, size_pitch_bytes_C, n * size_of_float, m, hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    hipblasDestroy(cublasHandle);
}
